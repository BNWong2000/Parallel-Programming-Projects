#include "hip/hip_runtime.h"
#include "thrust.h"
#include <stdio.h>
#include <chrono>


#define flatIndex(i, j, n) (j + (i * n))

struct squareRoot : public thrust::unary_function<float, float>{
    __host__ __device__
    float operator()(float input){
        return sqrtf(input);
    }
};

struct subAndSquare : public thrust::binary_function<float, float, float>{
    __host__ __device__
    float operator()(float x, float y){
        return (x-y) * (x-y);
    }
};

struct modByCount : public thrust::unary_function<int, int>{
    int num;

public:
    __host__ __device__ 
    modByCount(int x) : num(x) {}

    __host__ __device__
    int operator()(int input){
        return input % num;
    }
};

struct divByCount : public thrust::unary_function<int, int>{
    int num;

public:
    __host__ __device__ 
    divByCount(int x) : num(x) {}

    __host__ __device__
    int operator()(int input){
        return input / num;
    }
};

struct getPointInd : public thrust::unary_function<int, int>{
    int num;
    int dim; 

public:
    __host__ __device__ 
    getPointInd(int x, int y) : num(x), dim(y) {}

    __host__ __device__
    int operator()(int input){
        int pointIndex = input / num;
        int temp = (input % num) % dim; 
        return (pointIndex * dim) + temp;
    }
};

struct makePtrOffset : public thrust::unary_function<int, float>{
    floatIter initialPtr;

public:
    __host__ __device__ 
    makePtrOffset(floatIter x) : initialPtr(x) {}

    __host__ __device__
    float operator()(int input){
        return (*(initialPtr + input));
    }
};

typedef thrust::tuple<int, float> labelPair;
struct calcMin : public thrust::binary_function<labelPair, labelPair, labelPair>{
    __host__ __device__
    labelPair operator()(labelPair x, labelPair y){
        if(thrust::get<1>(x) < thrust::get<1>(y)){
            return x;
        }else{
            return y;
        }
    }
};

struct makeSumIndices : public thrust::unary_function<int, int>{
    intIter labelVec;
    int dimension;
public:
    __host__ __device__ 
    makeSumIndices(intIter x, int y) : labelVec(x), dimension(y) {}

    __host__ __device__
    int operator()(int input){
        int temp = input % dimension;
        int temp2 = input / dimension;
        return ((*(labelVec + temp2)) * dimension) + temp; 
    }
};

struct getAtIndex : public thrust::unary_function<int, float>{
    floatIter points;
public:
    __host__ __device__ 
    getAtIndex(floatIter x) : points(x) {}

    __host__ __device__
    float operator()(int input){
        return (*(points + input));
    }
};

struct doubleAdd : public thrust::binary_function<labelPair, labelPair, labelPair>{
    __host__ __device__
    labelPair operator()(labelPair x, labelPair y){
        return thrust::make_tuple(thrust::get<0>(x) + thrust::get<0>(y), thrust::get<1>(x) + thrust::get<1>(y));
    }
};

struct divFloatByInt : public thrust::binary_function<float, int, float>{
    __host__ __device__
    float operator()(float x, int y){
        return x / y;
    }
};

struct compareThreshold : public thrust::binary_function<float, double, bool>{
    __host__ __device__
    float operator()(float x, double y){
        return x < y;
    }
};

struct boolAnd : public thrust::binary_function<bool, bool, bool>{
    __host__ __device__
    bool operator()(bool x, bool y){
        return x && y;
    }
};

bool thrustConverged(   floatVec &oldCentroids,
                        floatVec &centroids,
                        int nCentroids,
                        int dimension,
                        double threshold ){
    // 
    floatVec diffSquares(nCentroids * dimension);

    thrust::transform(thrust::device, centroids.begin(), centroids.end(), oldCentroids.begin(), diffSquares.begin(), subAndSquare());
    intVec indices(nCentroids * dimension);
    thrust::sequence(thrust::device, indices.begin(), indices.end(), 0);
    thrust::transform(thrust::device, indices.begin(), indices.end(), indices.begin(), divByCount(dimension));

    intVec garbage(nCentroids * dimension);
    floatVec distances(nCentroids);

    thrust::reduce_by_key(thrust::device, indices.begin(), indices.end(), diffSquares.begin(), garbage.begin(), distances.begin());
    thrust::transform(thrust::device, distances.begin(), distances.end(), distances.begin(), squareRoot());
    thrust::device_vector<bool> isConverged(nCentroids);
    thrust::device_vector<double> thresholdVec(nCentroids);
    thrust::fill(thrust::device, thresholdVec.begin(), thresholdVec.end(), threshold);
    thrust::transform(thrust::device, distances.begin(), distances.end(), thresholdVec.begin(), isConverged.begin(), compareThreshold());
    
    bool result = thrust::reduce(thrust::device, isConverged.begin(), isConverged.end(), true, boolAnd());
    return result;
}

void thrustTakeAverage( floatVec &points, 
                        floatVec &centroids,
                        intVec &labels,
                        int nPoints,
                        int nCentroids,
                        int dimension ){
    intVec labelIndices(nPoints * dimension);

    intVec counts(nPoints * dimension);
    floatVec sums(nPoints * dimension);

    thrust::sequence(thrust::device, labelIndices.begin(), labelIndices.end(), 0);
    thrust::transform(thrust::device, labelIndices.begin(), labelIndices.end(), labelIndices.begin(), makeSumIndices(labels.begin(), dimension));

    thrust::copy(thrust::device, points.begin(), points.end(), sums.begin());

    thrust::stable_sort_by_key(thrust::device, labelIndices.begin(), labelIndices.end(), sums.begin(), thrust::less<int>());

    intVec outputLabels(nCentroids * dimension);

    floatVec outputFloats(nCentroids * dimension);
    intVec outputCounts(nCentroids * dimension);

    thrust::fill(thrust::device, counts.begin(), counts.end(), 1);
    thrust::reduce_by_key(thrust::device, labelIndices.begin(), labelIndices.end(),  sums.begin(), outputLabels.begin(), outputFloats.begin());

    thrust::reduce_by_key(thrust::device, labelIndices.begin(), labelIndices.end(), thrust::make_zip_iterator(thrust::make_tuple(counts.begin(), sums.begin())), outputLabels.begin(), thrust::make_zip_iterator(thrust::make_tuple(outputCounts.begin(), outputFloats.begin())), thrust::equal_to<int>(), doubleAdd());
    thrust::transform(thrust::device, outputFloats.begin(), outputFloats.end(), outputCounts.begin(), centroids.begin(), divFloatByInt());
}


void thrustFindNearestCentroid( floatVec &points, 
                                floatVec &centroids,
                                intVec &labels,
                                int nPoints,
                                int nCentroids,
                                int dimension ){
    // spread out points into centroids
    intVec pointIndices(nPoints * nCentroids * dimension);
    intVec centIndices(nPoints * nCentroids * dimension);

    floatVec centInputs(nPoints * nCentroids * dimension);
    floatVec pointInputs(nPoints * nCentroids * dimension);

    thrust::sequence(thrust::device, pointIndices.begin(), pointIndices.end(), 0);
    thrust::sequence(thrust::device, centIndices.begin(), centIndices.end(), 0);

    thrust::transform(thrust::device, pointIndices.begin(), pointIndices.end(), pointIndices.begin(), getPointInd(nCentroids * dimension, dimension));
    thrust::transform(thrust::device, centIndices.begin(), centIndices.end(), centIndices.begin(), modByCount(nCentroids * dimension));    

    thrust::transform(thrust::device, centIndices.begin(), centIndices.end(), centInputs.begin(), makePtrOffset(centroids.begin()));
    thrust::transform(thrust::device, pointIndices.begin(), pointIndices.end(), pointInputs.begin(), makePtrOffset(points.begin()));

    floatVec distancesIntermediate(nPoints * nCentroids * dimension);
    thrust::transform(thrust::device, pointInputs.begin(), pointInputs.end(), centInputs.begin(), distancesIntermediate.begin(), subAndSquare());

    // reset centIndices
    thrust::sequence(thrust::device, centIndices.begin(), centIndices.end(), 0);
    thrust::transform(thrust::device, centIndices.begin(), centIndices.end(), centIndices.begin(), divByCount(dimension)); 


    floatVec distances(nPoints * nCentroids);
    intVec distanceIndices(nPoints * nCentroids); 

    // We use centIndices as a key for a reduce by key
    thrust::reduce_by_key(thrust::device, centIndices.begin(), centIndices.end(), distancesIntermediate.begin(), distanceIndices.begin(), distances.begin() );
    thrust::transform(thrust::device, distances.begin(), distances.end(), distances.begin(), squareRoot());
    thrust::transform(thrust::device, distanceIndices.begin(), distanceIndices.end(), distanceIndices.begin(), modByCount(nCentroids));

    intVec tempKey(nPoints * nCentroids);
    thrust::sequence(thrust::device, tempKey.begin(), tempKey.end());
    thrust::transform(thrust::device, tempKey.begin(), tempKey.end(), tempKey.begin(), divByCount(nCentroids));
    thrust::reduce_by_key(thrust::device, tempKey.begin(), tempKey.end(), thrust::make_zip_iterator(thrust::make_tuple(distanceIndices.begin(),distances.begin())), pointIndices.begin(), thrust::make_zip_iterator(thrust::make_tuple(labels.begin(), distancesIntermediate.begin())), thrust::equal_to<int>(), calcMin());
}

void kmeans_cuda_thrust(args_t *args){
    thrust::device_vector<float> points(args->nVals * args->dimension);
    thrust::device_vector<float> centroids(args->num_cluster * args->dimension);
    thrust::device_vector<float> oldCentroids(args->num_cluster * args->dimension);
    thrust::device_vector<int> labels(args->nVals);

    // generate k random centroids,
    kmeans_srand(args->seed); // cmd_seed is a cmdline arg
    for (int i = 0; i < args->num_cluster; i++){
        int index = kmeans_rand() % args->nVals;
        for(int j = 0; j < args->dimension; j++){
            centroids[flatIndex(i,j,args->dimension)] = args->input_vals[index][j];
        }
    }

    for (int i = 0; i < args->nVals; i++){
        for(int j = 0; j < args->dimension; j++){
            points[flatIndex(i,j,args->dimension)] = args->input_vals[i][j];
        }
    }
    
    int iters = 0;
    bool done = false;
    const auto start = std::chrono::high_resolution_clock::now();
    while(!done){

        // copy centroids into old centroids
        thrust::copy(centroids.begin(), centroids.end(), oldCentroids.begin());
        iters++;

        // find nearest centroid
        thrustFindNearestCentroid(points, centroids, labels, args->nVals, args->num_cluster, args->dimension);

        thrust::fill(thrust::device, centroids.begin(), centroids.end(), 0.0f);

        thrustTakeAverage(points, centroids, labels, args->nVals, args->num_cluster, args->dimension);

        done = iters > args->max_num_iter || thrustConverged(oldCentroids, centroids, args->num_cluster, args->dimension, args->threshold);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> fp_ms = end - start;

    args->iters = iters;
    args->timeTaken = fp_ms.count()/iters;

    args->centroids = (float **)malloc(sizeof(float *) * args->num_cluster);
    for(int i = 0; i < args->num_cluster; i++){
        args->centroids[i] = (float *) malloc(sizeof(float) * args->dimension);
        for(int j = 0; j < args->dimension; j++){
            args->centroids[i][j] = centroids[flatIndex(i,j,args->dimension)];
        }
    }
    args->labels = (int *)malloc(sizeof(int) * args->nVals);
    for(int i = 0; i < args->nVals; i++){
        args->labels[i] = labels[i];
    }
}