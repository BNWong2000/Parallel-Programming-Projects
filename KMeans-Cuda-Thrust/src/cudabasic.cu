#include "hip/hip_runtime.h"
#include "cudabasic.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// #define TIME_MEMCPY

#define flatIndex(i, j, n) (j + (i * n))


/**
 * Executes a thread per point. 
 */
__global__ void cudaBasicFindNearestCentroids(  int *labels, 
                                                float *centroids, 
                                                float *points, 
                                                int nVals, 
                                                int nCentroids, 
                                                int dimension, 
                                                float minDist   ){
    // for each point compare with each centroid 

    // find index from thread/block id
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if(index < nVals){
        float closestDist = minDist;

        // iterate through centroids and compare.
        for(int i = 0; i < nCentroids; i++){

            // calculate euclidean distance. 
            float dist = 0.0f;
            for(int j = 0; j < dimension; j++){
                float temp = points[flatIndex(index, j, dimension)] - centroids[flatIndex(i, j, dimension)];
                temp *= temp;
                dist += temp;
            }
            dist = sqrtf(dist);

            if(dist < closestDist){
                closestDist = dist;
                labels[index] = i;
            }
        }
    }
}

__global__ void cudaBasicResetCentroidsToZero( float *centroids, int *nPointsPerCentroid, int nElements){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if(index < nElements){
        centroids[index] = 0.0f;
        nPointsPerCentroid[index] = 0;
    }
}

__global__ void cudaBasicSumPointsToCentroids(  int *labels,
                                                int *nPointsPerCentroid,
                                                float *centroids,
                                                float *points,
                                                int nCentroids,
                                                int nVals,
                                                int dimension ){
    
    // index is the point number. (1 thread per point)
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if(index < nVals){
        int labelIndex = labels[index];
        // Increment points per centroid
        atomicAdd(&nPointsPerCentroid[labelIndex], 1);
        // add the point to the centroid
        for(int i = 0; i < dimension; i++){
            atomicAdd(&centroids[flatIndex(labelIndex, i, dimension)], points[flatIndex(index, i, dimension)]);
        }
    }
}

__global__ void cudaBasicDivideCentroidsByCount(    int *nPointsPerCentroid,
                                                    float *centroids,
                                                    int nElements,
                                                    int dimension ){

    // index is every float in the centroids array (dimension * number of centroids).                                                 
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int centroidIndex = index / dimension;
    if(index < nElements && nPointsPerCentroid[centroidIndex] > 0){
        centroids[index] /= nPointsPerCentroid[centroidIndex];
    }
}

__global__ void cudaBasicConverged( float *centroids,
                                    float *oldCentroids,
                                    int nCentroids,
                                    int dimension,
                                    double threshold,
                                    int *notConvergedFlag ){

    // index is the centroid number. 
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if(index < nCentroids){
        float dist = 0.0f;
        for(int i = 0; i < dimension; i++){
            float temp = centroids[flatIndex(index, i, dimension)] - oldCentroids[flatIndex(index, i, dimension)];
            temp *= temp;
            dist += temp;
        }
        dist = sqrtf(dist);

        if(dist > threshold){
            atomicAdd(notConvergedFlag, 1);
        }
    }
}

void kmeans_cuda_basic(args_t *args){
    // TIMING:

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#ifdef TIME_MEMCPY
    hipEvent_t e2eStart, e2eStop;
    hipEventCreate(&e2eStart);
    hipEventCreate(&e2eStop);
    hipEventRecord(e2eStart);

#endif

    float *d_centroids;
    float *d_oldCentroids;
    float *d_points;
    int *labels;
    int *nPointsPerCentroid;
    int notConvergedFlag = 0;
    int *d_notConvergedFlag;
    hipMalloc((void **)&d_centroids, sizeof(float) * args->num_cluster * args->dimension);
    hipMalloc((void **)&d_oldCentroids, sizeof(float) * args->num_cluster * args->dimension);
    hipMalloc((void **)&d_points, sizeof(float) * args->nVals * args->dimension);
    hipMalloc((void **)&labels, sizeof(int) * args->nVals); // an array that maps to the index of the nearest centroid
    hipMalloc((void **)&nPointsPerCentroid, sizeof(int) * args->num_cluster);
    hipMalloc((void **)&d_notConvergedFlag, sizeof(int));

#ifdef TIME_MEMCPY
    float temp;
    float totalMemCpyTime = 0.0f;
    hipEvent_t memoryCopyStart, memoryCopyStop;
    hipEventCreate(&memoryCopyStart);
    hipEventCreate(&memoryCopyStop);
    hipEventRecord(memoryCopyStart);
#endif
    hipMemcpy(d_notConvergedFlag, &notConvergedFlag, sizeof(int), hipMemcpyHostToDevice);

#ifdef TIME_MEMCPY
    hipEventRecord(memoryCopyStop);
    hipEventSynchronize(memoryCopyStop);
    hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
    totalMemCpyTime += temp;
#endif

    float *tempCentroids = (float *)malloc(sizeof(float) * args->num_cluster * args->dimension);
    float *tempPoints = (float *)malloc(sizeof(float) * args->nVals * args->dimension);

    // generate k random centroids,
    kmeans_srand(args->seed); // cmd_seed is a cmdline arg
    for (int i=0; i < args->num_cluster; i++){
        int index = kmeans_rand() % args->nVals;
        memcpy(&tempCentroids[i * args->dimension], args->input_vals[index], args->dimension * sizeof(float));
    }

    for (int i = 0; i < args->nVals; i++){
        int index = i * args->dimension;
        memcpy(&tempPoints[index], args->input_vals[i], args->dimension * sizeof(float));
    }
#ifdef TIME_MEMCPY
    hipEventRecord(memoryCopyStart);
#endif
    hipMemcpy(d_centroids, tempCentroids, sizeof(float) * args->num_cluster * args->dimension, hipMemcpyHostToDevice);
    hipMemcpy(d_points, tempPoints, sizeof(float) * args->nVals * args->dimension, hipMemcpyHostToDevice);
#ifdef TIME_MEMCPY
    hipEventRecord(memoryCopyStop);
    hipEventSynchronize(memoryCopyStop);
    hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
    totalMemCpyTime += temp;
#endif
    free(tempCentroids);
    free(tempPoints);
    int iters = 0;
    bool done = false;

    int nThreadsPerBlock = 1024;

    // for kernels that need to run for every point. 
    // taking the integer ceiling.
    int nBlocksA = (args->nVals / nThreadsPerBlock) + ((args->nVals % nThreadsPerBlock) != 0);

    // for kernels that need to run for every dimension of every centroid
    int nElements = args->num_cluster * args->dimension;
    int nBlocksB = (nElements / nThreadsPerBlock) + ((nElements % nThreadsPerBlock) != 0);

    // for kernels that need to run for every centroid. 
    int nBlocksC = (args->num_cluster / nThreadsPerBlock) + ((args->num_cluster % nThreadsPerBlock) != 0);
    float totalKernalTime = 0.0f;

    hipEventRecord(start);

    while(!done){
        // Reset the notconverged flag
#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStart);
#endif

        hipMemcpy(d_notConvergedFlag, &notConvergedFlag, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_oldCentroids, d_centroids, sizeof(float) * args->num_cluster * args->dimension, hipMemcpyDeviceToDevice);

#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStop);
        hipEventSynchronize(memoryCopyStop);
        hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
        totalMemCpyTime += temp;
#endif
        iters++;
        
        // find nearest centroid
        
        cudaBasicFindNearestCentroids<<<nBlocksA,nThreadsPerBlock>>>(
                labels, 
                d_centroids, 
                d_points, 
                args->nVals, 
                args->num_cluster, 
                args->dimension, 
                std::numeric_limits<float>::max() );

        
        
        // Reset Centroids to zero, to take average. 
        cudaBasicResetCentroidsToZero<<<nBlocksB, nThreadsPerBlock>>>(d_centroids, nPointsPerCentroid, args->num_cluster * args->dimension);
        
        cudaBasicSumPointsToCentroids<<<nBlocksA, nThreadsPerBlock>>>(
                labels,
                nPointsPerCentroid,
                d_centroids,
                d_points,
                args->num_cluster,
                args->nVals,
                args->dimension );

        
        cudaBasicDivideCentroidsByCount<<<nBlocksB, nThreadsPerBlock>>>(
                nPointsPerCentroid,
                d_centroids,
                args->num_cluster * args->dimension,
                args->dimension );

        cudaBasicConverged<<<nBlocksC, nThreadsPerBlock>>>( d_centroids,
                                                            d_oldCentroids,
                                                            args->num_cluster,
                                                            args->dimension,
                                                            args->threshold,
                                                            d_notConvergedFlag );
#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStart);
#endif

        hipMemcpy(&notConvergedFlag, d_notConvergedFlag, sizeof(int), hipMemcpyDeviceToHost);
#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStop);
        hipEventSynchronize(memoryCopyStop);
        hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
        totalMemCpyTime += temp;
#endif

        done = iters > args->max_num_iter || (notConvergedFlag == 0); //args->max_num_iter
        notConvergedFlag = 0;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalKernalTime, start, stop);

    args->iters = iters;
    args->timeTaken = totalKernalTime/iters;

    args->centroids = (float **)malloc(sizeof(float *) * args->num_cluster);
    for(int i = 0; i < args->num_cluster; i++){
        args->centroids[i] = (float *) malloc(sizeof(float) * args->dimension);
        hipMemcpy(args->centroids[i], &d_centroids[args->dimension * i], sizeof(float) * args->dimension, hipMemcpyDeviceToHost);
    }
    args->labels = (int *)malloc(sizeof(int) * args->nVals);
    hipMemcpy(args->labels, labels, sizeof(int) * args->nVals, hipMemcpyDeviceToHost);

    
    hipFree(d_centroids);
    hipFree(d_oldCentroids);
    hipFree(d_points);
    hipFree(labels);
    hipFree(nPointsPerCentroid);
    hipFree(d_notConvergedFlag);
    // free everything here...


    hipEventDestroy(start);
    hipEventDestroy(stop);

    
#ifdef TIME_MEMCPY
    hipEventRecord(e2eStop);
    hipEventSynchronize(e2eStop);
    float totalTime;
    hipEventElapsedTime(&totalTime, e2eStart, e2eStop);
    printf("memcpy time: %lf, fractional time: %lf\n",totalMemCpyTime, (totalMemCpyTime)/totalTime);
    totalMemCpyTime /= iters;
#endif
}