#include "hip/hip_runtime.h"
#include "cudashmem.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// #define TIME_MEMCPY

#define flatIndex(i, j, n) (j + (i * n))


/**
 * Executes a thread per point. 
 */
__global__ void cudaShmemFindNearestCentroids(  int *labels, 
                                                float *centroids, 
                                                float *points, 
                                                int nVals, 
                                                int nCentroids, 
                                                int dimension, 
                                                float minDist   ){

    // All threads iterate through ALL centroids each time, so it makes sense to mark centroids as shared.                                                
    extern __shared__ float sCentroids[];

    // find index from thread/block id
    // This is a global index...
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    // If we have more centroids than threads, then we need to account for that when filling. 
    if(blockDim.x > nCentroids){
        int i = threadIdx.x;
        while(i < nCentroids){
            for(int j = 0; j < dimension; j++){
                sCentroids[flatIndex(i, j, dimension)] = centroids[flatIndex(i, j, dimension)];
            }
            i += blockDim.x;
        }
    }else{
        for(int i = 0; i < dimension; i++){
            sCentroids[flatIndex(threadIdx.x, i, dimension)] = centroids[flatIndex(threadIdx.x, i, dimension)];
        }
    }
    __syncthreads();

    if(index < nVals){
        float closestDist = minDist;

        // iterate through centroids and compare.
        for(int i = 0; i < nCentroids; i++){

            // calculate euclidean distance. 
            float dist = 0.0f;
            for(int j = 0; j < dimension; j++){
                float temp = points[flatIndex(index, j, dimension)] - sCentroids[flatIndex(i, j, dimension)];
                temp *= temp;
                dist += temp;
            }
            dist = sqrtf(dist);

            if(dist < closestDist){
                closestDist = dist;
                labels[index] = i;
            }
        }
    }
}

__global__ void cudaShmemResetCentroidsToZero( float *centroids, int *nPointsPerCentroid, int nElements){
    // It doesn't make sense to add shared memory any time here.
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if(index < nElements){
        centroids[index] = 0.0f;
        nPointsPerCentroid[index] = 0;
    }
}

__global__ void cudaShmemSumPointsToCentroids(  int *labels,
                                                int *nPointsPerCentroid,
                                                float *centroids,
                                                float *points,
                                                int nCentroids,
                                                int nVals,
                                                int dimension ){
    
    // index is the point number. (1 thread per point)
    extern __shared__ float sCentroids[];

    // find index from thread/block id
    // This is a global index...
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    // If we have more centroids than threads, then we need to account for that when filling. 
    if(blockDim.x > nCentroids){
        int i = threadIdx.x;
        while(i < nCentroids){
            for(int j = 0; j < dimension; j++){
                sCentroids[flatIndex(i, j, dimension)] = 0.0f;
            }
            i += blockDim.x;
        }
    }else{
        for(int i = 0; i < dimension; i++){
            sCentroids[flatIndex(threadIdx.x, i, dimension)] = 0.0f;
        }
    }
    __syncthreads();

    if(index < nVals){
        int labelIndex = labels[index];
        // Increment points per centroid
        atomicAdd(&nPointsPerCentroid[labelIndex], 1);
        // add the point to the centroid
        for(int i = 0; i < dimension; i++){
            atomicAdd(&sCentroids[flatIndex(labelIndex, i, dimension)], points[flatIndex(index, i, dimension)]);
        }
    }

    __syncthreads();
    if(blockDim.x > nCentroids){
        int i = threadIdx.x;
        while(i < nCentroids){
            for(int j = 0; j < dimension; j++){
                atomicAdd(&centroids[flatIndex(i, j, dimension)], sCentroids[flatIndex(i, j, dimension)]);
            }
            i += blockDim.x;
        }
    }else{
        for(int i = 0; i < dimension; i++){
            // Need to atomic add otherwise race condition.
            atomicAdd(&centroids[flatIndex(threadIdx.x, i, dimension)], sCentroids[flatIndex(threadIdx.x, i, dimension)]);
        }
    }
}

__global__ void cudaShmemDivideCentroidsByCount(    int *nPointsPerCentroid,
                                                    float *centroids,
                                                    int nElements,
                                                    int dimension ){

    // index is every float in the centroids array (dimension * number of centroids).                                                 
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int centroidIndex = index / dimension;
    if(index < nElements && nPointsPerCentroid[centroidIndex] > 0){
        centroids[index] /= nPointsPerCentroid[centroidIndex];
    }
}

__global__ void cudaShmemConverged( float *centroids,
                                    float *oldCentroids,
                                    int nCentroids,
                                    int dimension,
                                    double threshold,
                                    int *notConvergedFlag ){
    // index is the centroid number. 
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if(index < nCentroids){
        float dist = 0.0f;
        for(int i = 0; i < dimension; i++){
            float temp = centroids[flatIndex(index, i, dimension)] - oldCentroids[flatIndex(index, i, dimension)];
            temp *= temp;
            dist += temp;
        }
        dist = sqrtf(dist);
        if(dist > threshold){
            atomicAdd(notConvergedFlag, 1);
        }
    }
}

void kmeans_cuda_shmem(args_t *args){
    // TIMING:

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#ifdef TIME_MEMCPY
    hipEvent_t e2eStart, e2eStop;
    hipEventCreate(&e2eStart);
    hipEventCreate(&e2eStop);
    hipEventRecord(e2eStart);

#endif

    float *d_centroids;
    float *d_oldCentroids;
    float *d_points;
    int *labels;
    int *nPointsPerCentroid;
    int notConvergedFlag = 0;
    int *d_notConvergedFlag;

    hipMalloc((void **)&d_centroids, sizeof(float) * args->num_cluster * args->dimension);
    hipMalloc((void **)&d_oldCentroids, sizeof(float) * args->num_cluster * args->dimension);
    hipMalloc((void **)&d_points, sizeof(float) * args->nVals * args->dimension);
    hipMalloc((void **)&labels, sizeof(int) * args->nVals); // an array that maps to the index of the nearest centroid
    hipMalloc((void **)&nPointsPerCentroid, sizeof(int) * args->num_cluster);
    hipMalloc((void **)&d_notConvergedFlag, sizeof(int));

#ifdef TIME_MEMCPY
    float temp;
    float totalMemCpyTime = 0.0f;
    hipEvent_t memoryCopyStart, memoryCopyStop;
    hipEventCreate(&memoryCopyStart);
    hipEventCreate(&memoryCopyStop);
    hipEventRecord(memoryCopyStart);
#endif
    hipMemcpy(d_notConvergedFlag, &notConvergedFlag, sizeof(int), hipMemcpyHostToDevice);

#ifdef TIME_MEMCPY
    hipEventRecord(memoryCopyStop);
    hipEventSynchronize(memoryCopyStop);
    hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
    totalMemCpyTime += temp;
#endif

    float *tempCentroids = (float *)malloc(sizeof(float) * args->num_cluster * args->dimension);
    float *tempPoints = (float *)malloc(sizeof(float) * args->nVals * args->dimension);

    // generate k random centroids,
    kmeans_srand(args->seed); // cmd_seed is a cmdline arg
    for (int i=0; i < args->num_cluster; i++){
        int index = kmeans_rand() % args->nVals;
        memcpy(&tempCentroids[i * args->dimension], args->input_vals[index], args->dimension * sizeof(float));
    }

    for (int i = 0; i < args->nVals; i++){
        int index = i * args->dimension;
        memcpy(&tempPoints[index], args->input_vals[i], args->dimension * sizeof(float));
    }

#ifdef TIME_MEMCPY
    hipEventRecord(memoryCopyStart);
#endif

    hipMemcpy(d_centroids, tempCentroids, sizeof(float) * args->num_cluster * args->dimension, hipMemcpyHostToDevice);
    hipMemcpy(d_points, tempPoints, sizeof(float) * args->nVals * args->dimension, hipMemcpyHostToDevice);

#ifdef TIME_MEMCPY
    hipEventRecord(memoryCopyStop);
    hipEventSynchronize(memoryCopyStop);
    hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
    totalMemCpyTime += temp;
#endif

    free(tempCentroids);
    free(tempPoints);
    int iters = 0;
    bool done = false;

    int nThreadsPerBlock = 1024;

    // for kernels that need to run for every point. 
    // taking the integer ceiling.
    int nBlocksA = (args->nVals / nThreadsPerBlock) + ((args->nVals % nThreadsPerBlock) != 0);

    // for kernels that need to run for every dimension of every centroid
    int nElements = args->num_cluster * args->dimension;
    int nBlocksB = (nElements / nThreadsPerBlock) + ((nElements % nThreadsPerBlock) != 0);

    // for kernels that need to run for every centroid. 
    int nBlocksC = (args->num_cluster / nThreadsPerBlock) + ((args->num_cluster % nThreadsPerBlock) != 0);
    float totalKernalTime = 0.0f;
    hipEventRecord(start);
    while(!done){
#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStart);
#endif
        // Reset the notconverged flag
        hipMemcpy(d_notConvergedFlag, &notConvergedFlag, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_oldCentroids, d_centroids, sizeof(float) * args->num_cluster * args->dimension, hipMemcpyDeviceToDevice);
        // cudaBasicCopyCentroids<<<nBlocks, nThreads>>> (d_oldCentroids, d_centroids,sizeof(float) * args->num_cluster * args->dimension);

#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStop);
        hipEventSynchronize(memoryCopyStop);
        hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
        totalMemCpyTime += temp;
#endif
        iters++;
        // find nearest centroid
        
        cudaShmemFindNearestCentroids<<<nBlocksA, nThreadsPerBlock, args->num_cluster * args->dimension * sizeof(float)>>>(
                labels, 
                d_centroids, 
                d_points, 
                args->nVals, 
                args->num_cluster, 
                args->dimension, 
                std::numeric_limits<float>::max() );

        
        
        // Reset Centroids to zero, to take average. 
        cudaShmemResetCentroidsToZero<<<nBlocksB, nThreadsPerBlock>>>(d_centroids, nPointsPerCentroid, args->num_cluster * args->dimension);
        
        cudaShmemSumPointsToCentroids<<<nBlocksA, nThreadsPerBlock, args->num_cluster * args->dimension * sizeof(float)>>>(
                labels,
                nPointsPerCentroid,
                d_centroids,
                d_points,
                args->num_cluster,
                args->nVals,
                args->dimension );
        
        cudaShmemDivideCentroidsByCount<<<nBlocksB, nThreadsPerBlock>>>(
                nPointsPerCentroid,
                d_centroids,
                args->num_cluster * args->dimension,
                args->dimension );

        cudaShmemConverged<<<nBlocksC, nThreadsPerBlock>>>( d_centroids,
                                                            d_oldCentroids,
                                                            args->num_cluster,
                                                            args->dimension,
                                                            args->threshold,
                                                            d_notConvergedFlag );
#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStart);
#endif
        hipMemcpy(&notConvergedFlag, d_notConvergedFlag, sizeof(int), hipMemcpyDeviceToHost);

#ifdef TIME_MEMCPY
        hipEventRecord(memoryCopyStop);
        hipEventSynchronize(memoryCopyStop);
        hipEventElapsedTime(&temp, memoryCopyStart, memoryCopyStop);
        totalMemCpyTime += temp;
#endif

        done = iters > args->max_num_iter || (notConvergedFlag == 0); //args->max_num_iter
        notConvergedFlag = 0;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalKernalTime, start, stop);
    // std::cout << "time is "<< totalKernalTime/iters << "ms" << std::endl;

    args->iters = iters;
    args->timeTaken = totalKernalTime/iters;

    args->centroids = (float **)malloc(sizeof(float *) * args->num_cluster);
    for(int i = 0; i < args->num_cluster; i++){
        args->centroids[i] = (float *) malloc(sizeof(float) * args->dimension);
        hipMemcpy(args->centroids[i], &d_centroids[args->dimension * i], sizeof(float) * args->dimension, hipMemcpyDeviceToHost);
    }

    args->labels = (int *)malloc(sizeof(int) * args->nVals);
    hipMemcpy(args->labels, labels, sizeof(int) * args->nVals, hipMemcpyDeviceToHost);
    
    hipFree(d_centroids);
    hipFree(d_oldCentroids);
    hipFree(d_points);
    hipFree(labels);
    hipFree(nPointsPerCentroid);
    hipFree(d_notConvergedFlag);
    // free everything here...


    hipEventDestroy(start);
    hipEventDestroy(stop);
#ifdef TIME_MEMCPY
    hipEventRecord(e2eStop);
    hipEventSynchronize(e2eStop);
    float totalTime;
    hipEventElapsedTime(&totalTime, e2eStart, e2eStop);
    printf("memcpy time: %lf, fractional time: %lf\n",totalMemCpyTime, (totalMemCpyTime)/totalTime);
    totalMemCpyTime /= iters;
#endif
}